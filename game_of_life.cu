#include "hip/hip_runtime.h"
#include "game_of_life.h"

#include <stdio.h>

__device__ int countAliveNeis(int idx, bool *state, int size)
{
    int result = 0;
    int row = idx / size;
    int col = idx % size;

    // Neighbor offsets for periodic boundary conditions (wrapping grid edges)
    const int neighborOffsets[8][2] = {
        {-1, -1}, {-1, 0}, {-1, 1}, {0, -1}, {0, 1}, {1, -1}, {1, 0}, {1, 1}};

    for (int i = 0; i < 8; i++)
    {
        int nRow = (row + neighborOffsets[i][0] + size) % size; // Wrap rows
        int nCol = (col + neighborOffsets[i][1] + size) % size; // Wrap columns
        int neighborIdx = nRow * size + nCol;
        result += state[neighborIdx];
    }

    return result;
}

__global__ void computeGameOfLifeStep(bool *currState, bool *nextState, int size)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < size && col < size)
    {
        int idx = row * size + col;
        int aliveNeis = countAliveNeis(idx, currState, size);
        if (currState[idx] == 0 && aliveNeis == 3)
            nextState[idx] = 1;
        else if (currState[idx] == 1 && (aliveNeis == 2 || aliveNeis == 3))
            nextState[idx] = 1;
        else
            nextState[idx] = 0;
    }
}

void calculateGameOfLife(bool *hostState, int size, int steps, bool usePinned)
{
    bool *deviceState;
    // Allocate memory according to usePinned flag
    if (usePinned)
    {
        if (hipHostMalloc(&deviceState, size * size * sizeof(bool)) != hipSuccess)
        {
            fprintf(stderr, "Error: Failed to allocate pinned memory for CUDA");
        }
    }
    else if (hipMalloc(&deviceState, size * size * sizeof(bool)) != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate memory for CUDA");
    }

    // Copy memory from host to device
    if (hipMemcpy(deviceState, hostState, size * size * sizeof(bool), hipMemcpyHostToDevice) != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from host to CUDA");
    }

    bool *deviceTempState;
    // Allocate memory according to usePinned flag
    if (usePinned)
    {
        if (hipHostMalloc(&deviceTempState, size * size * sizeof(bool)) != hipSuccess)
        {
            fprintf(stderr, "Error: Failed to allocate pinned memory for CUDA");
        }
    }
    else if (hipMalloc(&deviceTempState, size * size * sizeof(bool)) != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to allocate memory for CUDA");
    }

    // Copy memory from host to device
    if (hipMemset(deviceTempState, 0, size * size * sizeof(bool)) != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to set memory for CUDA");
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 blocks((size + 15) / 16, (size + 15) / 16);

    for (int i = 0; i < steps; ++i)
    {
        computeGameOfLifeStep<<<blocks, threadsPerBlock>>>(deviceState, deviceTempState, size);
        hipDeviceSynchronize();

        bool *temp = deviceState;
        deviceState = deviceTempState;
        deviceTempState = temp;
    }

    if (hipMemcpy(hostState, deviceState, size * size * sizeof(bool), hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Error: Failed to copy memory from CUDA to host");
        return;
    }

    if (usePinned)
        hipHostFree(deviceTempState);
    else
        hipFree(deviceTempState);
    if (usePinned)
        hipHostFree(deviceState);
    else
        hipFree(deviceState);
}